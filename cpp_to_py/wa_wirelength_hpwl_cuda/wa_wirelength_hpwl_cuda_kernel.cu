#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>
#include <vector>

template <typename scalar_t>
__global__ void node_pos_to_pin_pos_cuda_forward_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> node_pos,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> pin_id2node_id,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_pos,
    int num_pins) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // pin index
    if (i < num_pins) {
        const int c = index & 1;  // channel index
        int64_t node_id = pin_id2node_id[i];
        pin_pos[i][c] += node_pos[node_id][c];
    }
}

template <typename scalar_t>
__global__ void masked_scale_hpwl_cuda_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_pos,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list_end,
    const torch::PackedTensorAccessor32<bool, 1, torch::RestrictPtrTraits> net_mask,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> hpwl_scale,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> partial_hpwl,
    int num_nets) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // net index
    if (i < num_nets && net_mask[i]) {
        const int c = index & 1;  // channel index
        int64_t start_idx = 0;
        if (i != 0) {
            start_idx = hyperedge_list_end[i - 1];
        }
        int64_t end_idx = hyperedge_list_end[i];
        if (end_idx != start_idx) {
            int64_t pin_id = hyperedge_list[start_idx];
            scalar_t x_min = pin_pos[pin_id][c];
            scalar_t x_max = pin_pos[pin_id][c];
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                x_min = min(xx, x_min);
                x_max = max(xx, x_max);
            }
            partial_hpwl[i][c] = round(abs(x_max - x_min) * hpwl_scale[c]);
        }
    }
}

template <typename scalar_t>
__global__ void wa_wirelength_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_pos,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list_end,
    const torch::PackedTensorAccessor32<bool, 1, torch::RestrictPtrTraits> net_mask,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> partial_wa_wl,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_grad,
    int num_nets,
    float inv_gamma) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // net index
    if (i < num_nets && net_mask[i]) {
        const int c = index & 1;  // channel index
        int64_t start_idx = 0;
        if (i != 0) {
            start_idx = hyperedge_list_end[i - 1];
        }
        int64_t end_idx = hyperedge_list_end[i];
        if (end_idx != start_idx) {
            int64_t pin_id = hyperedge_list[start_idx];
            scalar_t x_min = pin_pos[pin_id][c];
            scalar_t x_max = pin_pos[pin_id][c];
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                x_min = min(xx, x_min);
                x_max = max(xx, x_max);
            }

            scalar_t xexp_x_sum = 0;
            scalar_t xexp_nx_sum = 0;
            scalar_t exp_x_sum = 0;
            scalar_t exp_nx_sum = 0;

            for (int64_t idx = start_idx; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                scalar_t exp_x = exp((xx - x_max) * inv_gamma);
                scalar_t exp_nx = exp((x_min - xx) * inv_gamma);

                xexp_x_sum += xx * exp_x;
                xexp_nx_sum += xx * exp_nx;
                exp_x_sum += exp_x;
                exp_nx_sum += exp_nx;
            }

            scalar_t wl = xexp_x_sum / exp_x_sum - xexp_nx_sum / exp_nx_sum;
            partial_wa_wl[i][c] = wl;

            scalar_t b_x = inv_gamma / (exp_x_sum);
            scalar_t a_x = (1.0 - b_x * xexp_x_sum) / exp_x_sum;
            scalar_t b_nx = -inv_gamma / (exp_nx_sum);
            scalar_t a_nx = (1.0 - b_nx * xexp_nx_sum) / exp_nx_sum;

            for (int64_t idx = start_idx; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                scalar_t exp_x = exp((xx - x_max) * inv_gamma);
                scalar_t exp_nx = exp((x_min - xx) * inv_gamma);

                pin_grad[hyperedge_list[idx]][c] = (a_x + b_x * xx) * exp_x - (a_nx + b_nx * xx) * exp_nx;
            }
        }
    }
}

template <typename scalar_t>
__global__ void wa_wirelength_hpwl_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_pos,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list_end,
    const torch::PackedTensorAccessor32<bool, 1, torch::RestrictPtrTraits> net_mask,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> partial_wa_wl,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> partial_hpwl,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_grad,
    int num_nets,
    float inv_gamma) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // net index
    if (i < num_nets && net_mask[i]) {
        const int c = index & 1;  // channel index
        int64_t start_idx = 0;
        if (i != 0) {
            start_idx = hyperedge_list_end[i - 1];
        }
        int64_t end_idx = hyperedge_list_end[i];
        if (end_idx != start_idx) {
            int64_t pin_id = hyperedge_list[start_idx];
            scalar_t x_min = pin_pos[pin_id][c];
            scalar_t x_max = pin_pos[pin_id][c];
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                x_min = min(xx, x_min);
                x_max = max(xx, x_max);
            }
            partial_hpwl[i][c] = abs(x_max - x_min);

            scalar_t xexp_x_sum = 0;
            scalar_t xexp_nx_sum = 0;
            scalar_t exp_x_sum = 0;
            scalar_t exp_nx_sum = 0;

            for (int64_t idx = start_idx; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                scalar_t exp_x = exp((xx - x_max) * inv_gamma);
                scalar_t exp_nx = exp((x_min - xx) * inv_gamma);

                xexp_x_sum += xx * exp_x;
                xexp_nx_sum += xx * exp_nx;
                exp_x_sum += exp_x;
                exp_nx_sum += exp_nx;
            }

            scalar_t wl = xexp_x_sum / exp_x_sum - xexp_nx_sum / exp_nx_sum;
            partial_wa_wl[i][c] = wl;

            scalar_t b_x = inv_gamma / (exp_x_sum);
            scalar_t a_x = (1.0 - b_x * xexp_x_sum) / exp_x_sum;
            scalar_t b_nx = -inv_gamma / (exp_nx_sum);
            scalar_t a_nx = (1.0 - b_nx * xexp_nx_sum) / exp_nx_sum;

            for (int64_t idx = start_idx; idx < end_idx; idx++) {
                scalar_t xx = pin_pos[hyperedge_list[idx]][c];
                scalar_t exp_x = exp((xx - x_max) * inv_gamma);
                scalar_t exp_nx = exp((x_min - xx) * inv_gamma);

                pin_grad[hyperedge_list[idx]][c] = (a_x + b_x * xx) * exp_x - (a_nx + b_nx * xx) * exp_nx;
            }
        }
    }
}

template <typename scalar_t>
__global__ void wa_wirelength_masked_scale_hpwl_kernel(
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_pos,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list,
    const torch::PackedTensorAccessor32<int64_t, 1, torch::RestrictPtrTraits> hyperedge_list_end,
    const torch::PackedTensorAccessor32<bool, 1, torch::RestrictPtrTraits> net_mask,
    const torch::PackedTensorAccessor32<scalar_t, 1, torch::RestrictPtrTraits> hpwl_scale,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> partial_wa_wl,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> partial_hpwl,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> pin_grad,
    int num_nets,
    float inv_gamma) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int i = index >> 1;  // net index
    if (i < num_nets && net_mask[i]) {
        const int c = index & 1;  // channel index
        int64_t start_idx = 0;
        if (i != 0) {
            start_idx = hyperedge_list_end[i - 1];
        }
        int64_t end_idx = hyperedge_list_end[i];
        if (end_idx != start_idx) {
            int64_t pin_id = hyperedge_list[start_idx];
            scalar_t x_min = pin_pos[pin_id][c];
            scalar_t x_max = pin_pos[pin_id][c];
            for (int64_t idx = start_idx + 1; idx < end_idx; idx++) {
                scalar_t cur_x = pin_pos[hyperedge_list[idx]][c];
                x_min = min(cur_x, x_min);
                x_max = max(cur_x, x_max);
            }
            partial_hpwl[i][c] = round((x_max - x_min) * hpwl_scale[c]);
            // partial_hpwl[i][c] = round(abs(x_max - x_min) * hpwl_scale[c]);

            scalar_t sum_x_exp_x = 0;
            scalar_t sum_x_exp_nx = 0;
            scalar_t sum_exp_x = 0;
            scalar_t sum_exp_nx = 0;

            for (int64_t idx = start_idx; idx < end_idx; idx++) {
                scalar_t cur_x = pin_pos[hyperedge_list[idx]][c];
                scalar_t recenter_exp_x = exp((cur_x - x_max) * inv_gamma);
                scalar_t recenter_exp_nx = exp((x_min - cur_x) * inv_gamma);

                sum_x_exp_x += cur_x * recenter_exp_x;
                sum_x_exp_nx += cur_x * recenter_exp_nx;
                sum_exp_x += recenter_exp_x;
                sum_exp_nx += recenter_exp_nx;
            }
            scalar_t inv_sum_exp_x = 1 / sum_exp_x;
            scalar_t inv_sum_exp_nx = 1 / sum_exp_nx;

            scalar_t s_x = sum_x_exp_x * inv_sum_exp_x;
            scalar_t ns_nx = sum_x_exp_nx * inv_sum_exp_nx;
            partial_wa_wl[i][c] = s_x - ns_nx;
            scalar_t x_coeff = inv_gamma * inv_sum_exp_x;
            scalar_t nx_coeff = -inv_gamma * inv_sum_exp_nx;
            scalar_t grad_const = (1 - inv_gamma * s_x) * inv_sum_exp_x;
            scalar_t grad_nconst = (1 + inv_gamma * ns_nx) * inv_sum_exp_nx;

            for (int64_t idx = start_idx; idx < end_idx; idx++) {
                int64_t pin_id = hyperedge_list[idx];
                scalar_t cur_x = pin_pos[pin_id][c];
                scalar_t recenter_exp_x = exp((cur_x - x_max) * inv_gamma);
                scalar_t recenter_exp_nx = exp((x_min - cur_x) * inv_gamma);

                pin_grad[pin_id][c] = (grad_const + x_coeff * cur_x) * recenter_exp_x -
                                      (grad_nconst + nx_coeff * cur_x) * recenter_exp_nx;
            }
        }
    }
}

std::vector<torch::Tensor> wa_wirelength_cuda(torch::Tensor node_pos,
                                              torch::Tensor pin_id2node_id,
                                              torch::Tensor pin_rel_cpos,
                                              torch::Tensor hyperedge_list,
                                              torch::Tensor hyperedge_list_end,
                                              torch::Tensor net_mask,
                                              float gamma) {
    hipSetDevice(node_pos.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const auto num_nodes = node_pos.size(0);
    const auto num_pins = pin_id2node_id.size(0);
    const auto num_nets = hyperedge_list_end.size(0);
    const auto num_channels = 2;  // x, y

    auto pin_pos = pin_rel_cpos.clone();  // pin
    auto partial_wa_wl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto pin_grad = torch::zeros({num_pins, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));

    const int threads = 128;
    const int blocks = (num_pins * 2 + threads - 1) / threads;

    AT_DISPATCH_ALL_TYPES(node_pos.scalar_type(), "node_pos_to_pin_pos_cuda_forward", ([&] {
                              node_pos_to_pin_pos_cuda_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                                  node_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_id2node_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_pins);
                          }));

    const int threads2 = 128;
    const int blocks2 = (num_nets * 2 + threads2 - 1) / threads2;

    float inv_gamma = 1 / gamma;
    AT_DISPATCH_ALL_TYPES(pin_pos.scalar_type(), "wa_wirelength", ([&] {
                              wa_wirelength_kernel<scalar_t><<<blocks2, threads2, 0, stream>>>(
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  hyperedge_list.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  hyperedge_list_end.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  net_mask.packed_accessor32<bool, 1, torch::RestrictPtrTraits>(),
                                  partial_wa_wl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_grad.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_nets,
                                  inv_gamma);
                          }));

    auto node_grad = torch::zeros({num_nodes, num_channels}, torch::dtype(pin_grad.dtype()).device(pin_grad.device()));
    const auto pin_id2node_id_view = pin_id2node_id.unsqueeze(1).expand({-1, 2});
    node_grad.scatter_add_(0, pin_id2node_id_view, pin_grad);

    return {partial_wa_wl, node_grad};
}

torch::Tensor masked_scale_hpwl_sum_cuda(torch::Tensor node_pos,
                                         torch::Tensor pin_id2node_id,
                                         torch::Tensor pin_rel_cpos,
                                         torch::Tensor hyperedge_list,
                                         torch::Tensor hyperedge_list_end,
                                         torch::Tensor net_mask,
                                         torch::Tensor hpwl_scale) {
    hipSetDevice(node_pos.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const auto num_nodes = node_pos.size(0);
    const auto num_pins = pin_id2node_id.size(0);
    const auto num_nets = hyperedge_list_end.size(0);
    const auto num_channels = 2;  // x, y

    auto pin_pos = pin_rel_cpos.clone();  // pin
    auto partial_hpwl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));

    const int threads = 128;
    const int blocks = (num_pins * 2 + threads - 1) / threads;

    AT_DISPATCH_ALL_TYPES(node_pos.scalar_type(), "node_pos_to_pin_pos_cuda_forward", ([&] {
                              node_pos_to_pin_pos_cuda_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                                  node_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_id2node_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_pins);
                          }));

    const int threads2 = 128;
    const int blocks2 = (num_nets * 2 + threads2 - 1) / threads2;

    AT_DISPATCH_ALL_TYPES(pin_pos.scalar_type(), "masked_scale_hpwl_cuda", ([&] {
                              masked_scale_hpwl_cuda_kernel<scalar_t><<<blocks2, threads2, 0, stream>>>(
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  hyperedge_list.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  hyperedge_list_end.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  net_mask.packed_accessor32<bool, 1, torch::RestrictPtrTraits>(),
                                  hpwl_scale.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                  partial_hpwl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_nets);
                          }));
    const auto total_hpwl = partial_hpwl.sum();

    return total_hpwl;
}

std::vector<torch::Tensor> wa_wirelength_hpwl_cuda(torch::Tensor node_pos,
                                                   torch::Tensor pin_id2node_id,
                                                   torch::Tensor pin_rel_cpos,
                                                   torch::Tensor hyperedge_list,
                                                   torch::Tensor hyperedge_list_end,
                                                   torch::Tensor net_mask,
                                                   float gamma) {
    hipSetDevice(node_pos.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const auto num_nodes = node_pos.size(0);
    const auto num_pins = pin_id2node_id.size(0);
    const auto num_nets = hyperedge_list_end.size(0);
    const auto num_channels = 2;  // x, y

    auto pin_pos = pin_rel_cpos.clone();  // pin
    auto partial_wa_wl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto partial_hpwl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto pin_grad = torch::zeros({num_pins, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));

    const int threads = 128;
    const int blocks = (num_pins * 2 + threads - 1) / threads;

    AT_DISPATCH_ALL_TYPES(node_pos.scalar_type(), "node_pos_to_pin_pos_cuda_forward", ([&] {
                              node_pos_to_pin_pos_cuda_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                                  node_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_id2node_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_pins);
                          }));

    const int threads2 = 128;
    const int blocks2 = (num_nets * 2 + threads2 - 1) / threads2;

    float inv_gamma = 1 / gamma;
    AT_DISPATCH_ALL_TYPES(pin_pos.scalar_type(), "wa_wirelength_hpwl", ([&] {
                              wa_wirelength_hpwl_kernel<scalar_t><<<blocks2, threads2, 0, stream>>>(
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  hyperedge_list.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  hyperedge_list_end.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  net_mask.packed_accessor32<bool, 1, torch::RestrictPtrTraits>(),
                                  partial_wa_wl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  partial_hpwl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_grad.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_nets,
                                  inv_gamma);
                          }));

    auto node_grad = torch::zeros({num_nodes, num_channels}, torch::dtype(pin_grad.dtype()).device(pin_grad.device()));
    const auto pin_id2node_id_view = pin_id2node_id.unsqueeze(1).expand({-1, 2});
    node_grad.scatter_add_(0, pin_id2node_id_view, pin_grad);

    return {partial_wa_wl, node_grad, partial_hpwl};
}

std::vector<torch::Tensor> wa_wirelength_masked_scale_hpwl_cuda(torch::Tensor node_pos,
                                                                torch::Tensor pin_id2node_id,
                                                                torch::Tensor pin_rel_cpos,
                                                                torch::Tensor hyperedge_list,
                                                                torch::Tensor hyperedge_list_end,
                                                                torch::Tensor net_mask,
                                                                torch::Tensor hpwl_scale,
                                                                float gamma) {
    hipSetDevice(node_pos.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const auto num_nodes = node_pos.size(0);
    const auto num_pins = pin_id2node_id.size(0);
    const auto num_nets = hyperedge_list_end.size(0);
    const auto num_channels = 2;  // x, y

    auto pin_pos = pin_rel_cpos.clone();  // pin
    auto partial_wa_wl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto partial_hpwl = torch::zeros({num_nets, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));
    auto pin_grad = torch::zeros({num_pins, num_channels}, torch::dtype(pin_pos.dtype()).device(pin_pos.device()));

    const int threads = 128;
    const int blocks = (num_pins * 2 + threads - 1) / threads;

    AT_DISPATCH_ALL_TYPES(node_pos.scalar_type(), "node_pos_to_pin_pos_cuda_forward", ([&] {
                              node_pos_to_pin_pos_cuda_forward_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
                                  node_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_id2node_id.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_pins);
                          }));

    const int threads2 = 128;
    const int blocks2 = (num_nets * 2 + threads2 - 1) / threads2;

    float inv_gamma = 1 / gamma;
    AT_DISPATCH_ALL_TYPES(pin_pos.scalar_type(), "wa_wirelength_masked_scale_hpwl", ([&] {
                              wa_wirelength_masked_scale_hpwl_kernel<scalar_t><<<blocks2, threads2, 0, stream>>>(
                                  pin_pos.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  hyperedge_list.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  hyperedge_list_end.packed_accessor32<int64_t, 1, torch::RestrictPtrTraits>(),
                                  net_mask.packed_accessor32<bool, 1, torch::RestrictPtrTraits>(),
                                  hpwl_scale.packed_accessor32<scalar_t, 1, torch::RestrictPtrTraits>(),
                                  partial_wa_wl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  partial_hpwl.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  pin_grad.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
                                  num_nets,
                                  inv_gamma);
                          }));

    auto node_grad = torch::zeros({num_nodes, num_channels}, torch::dtype(pin_grad.dtype()).device(pin_grad.device()));
    const auto pin_id2node_id_view = pin_id2node_id.unsqueeze(1).expand({-1, 2});
    node_grad.scatter_add_(0, pin_id2node_id_view, pin_grad);

    return {partial_wa_wl, node_grad, partial_hpwl};
}