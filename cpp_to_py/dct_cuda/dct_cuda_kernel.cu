#include "hip/hip_runtime.h"
/**
 * @file   dct2_fft2_cuda_kernel.cu
 * @author Zixuan Jiang, Jiaqi Gu
 * @date   Apr 2019
 * @brief  Refernece: Byeong Lee, "A new algorithm to compute the discrete cosine Transform,"
 *      in IEEE Transactions on Acoustics, Speech, and Signal Processing, vol. 32, no. 6, pp. 1243-1245, December 1984.
 *      The preprocess and postprocess of 2d dct and 2d idct are discussed in the original paper.
 *      idct(idxst(x)) and idxst(idct(x)) are similar to the idct2d(x),
 *      except tiny modifications on preprocessing and postprocessing
 */

#include <ATen/cuda/HIPContext.h>
#include <float.h>
#include <math.h>
#include <torch/extension.h>

#include "hip/hip_runtime.h"

// ComplexType
template <typename T>
struct ComplexType {
    T x;
    T y;
    __host__ __device__ ComplexType() {
        x = 0;
        y = 0;
    }

    __host__ __device__ ComplexType(T real, T imag) {
        x = real;
        y = imag;
    }

    __host__ __device__ ~ComplexType() {}
};

template <typename T>
inline __host__ __device__ ComplexType<T> complexMul(const ComplexType<T> &x, const ComplexType<T> &y) {
    ComplexType<T> res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = x.x * y.y + x.y * y.x;
    return res;
}

template <typename T>
inline __host__ __device__ T RealPartOfMul(const ComplexType<T> &x, const ComplexType<T> &y) {
    return x.x * y.x - x.y * y.y;
}

template <typename T>
inline __host__ __device__ T ImaginaryPartOfMul(const ComplexType<T> &x, const ComplexType<T> &y) {
    return x.x * y.y + x.y * y.x;
}

template <typename T>
inline __host__ __device__ ComplexType<T> complexAdd(const ComplexType<T> &x, const ComplexType<T> &y) {
    ComplexType<T> res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

template <typename T>
inline __host__ __device__ ComplexType<T> complexSubtract(const ComplexType<T> &x, const ComplexType<T> &y) {
    ComplexType<T> res;
    res.x = x.x - y.x;
    res.y = x.y - y.y;
    return res;
}

template <typename T>
inline __host__ __device__ ComplexType<T> complexConj(const ComplexType<T> &x) {
    ComplexType<T> res;
    res.x = x.x;
    res.y = -x.y;
    return res;
}

template <typename T>
inline __host__ __device__ ComplexType<T> complexMulConj(const ComplexType<T> &x, const ComplexType<T> &y) {
    ComplexType<T> res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = -(x.x * y.y + x.y * y.x);
    return res;
}

#define TPB (16)

inline __device__ int INDEX(const int hid, const int wid, const int N) { return (hid * N + wid); }

// dct2_fft2
template <typename T>
__global__ void dct2dPreprocess(const T *x, T *y, const int M, const int N, const int halfN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N) {
        int index;
        int cond = (((hid & 1) == 0) << 1) | ((wid & 1) == 0);
        switch (cond) {
            case 0:
                index = INDEX(2 * M - (hid + 1), N - (wid + 1) / 2, halfN);
                break;
            case 1:
                index = INDEX(2 * M - (hid + 1), wid / 2, halfN);
                break;
            case 2:
                index = INDEX(hid, N - (wid + 1) / 2, halfN);
                break;
            case 3:
                index = INDEX(hid, wid / 2, halfN);
                break;
            default:
                break;
        }
        y[index] = x[INDEX(hid, wid, N)];
    }
}

template <typename T>
void dct2dPreprocessCudaLauncher(const T *x, T *y, const int M, const int N) {
    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    dct2dPreprocess<T><<<gridSize, blockSize, 0, stream>>>(x, y, M, N, N / 2);
}

template <typename T, typename TComplex>
__global__ void __launch_bounds__(TPB *TPB, 4) dct2dPostprocess(const TComplex *V,
                                                                T *y,
                                                                const int M,
                                                                const int N,
                                                                const int halfM,
                                                                const int halfN,
                                                                const T two_over_MN,
                                                                const T four_over_MN,
                                                                const TComplex *__restrict__ expkM,
                                                                const TComplex *__restrict__ expkN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < halfM && wid < halfN) {
        int cond = ((hid != 0) << 1) | (wid != 0);
        switch (cond) {
            case 0: {
                y[0] = V[0].x * four_over_MN;
                y[halfN] = RealPartOfMul(expkN[halfN], V[halfN]) * four_over_MN;
                y[INDEX(halfM, 0, N)] = expkM[halfM].x * V[INDEX(halfM, 0, halfN + 1)].x * four_over_MN;
                y[INDEX(halfM, halfN, N)] =
                    expkM[halfM].x * RealPartOfMul(expkN[halfN], V[INDEX(halfM, halfN, halfN + 1)]) * four_over_MN;
                break;
            }

            case 1: {
                ComplexType<T> tmp;

                tmp = V[wid];
                y[wid] = RealPartOfMul(expkN[wid], tmp) * four_over_MN;
                y[N - wid] = -ImaginaryPartOfMul(expkN[wid], tmp) * four_over_MN;

                tmp = V[INDEX(halfM, wid, halfN + 1)];
                y[INDEX(halfM, wid, N)] = expkM[halfM].x * RealPartOfMul(expkN[wid], tmp) * four_over_MN;
                y[INDEX(halfM, N - wid, N)] = -expkM[halfM].x * ImaginaryPartOfMul(expkN[wid], tmp) * four_over_MN;
                break;
            }

            case 2: {
                ComplexType<T> tmp1, tmp2, tmp_up, tmp_down;
                tmp1 = V[INDEX(hid, 0, halfN + 1)];
                tmp2 = V[INDEX(M - hid, 0, halfN + 1)];
                tmp_up.x = expkM[hid].x * (tmp1.x + tmp2.x) + expkM[hid].y * (tmp2.y - tmp1.y);
                tmp_down.x = -expkM[hid].y * (tmp1.x + tmp2.x) + expkM[hid].x * (tmp2.y - tmp1.y);
                y[INDEX(hid, 0, N)] = tmp_up.x * two_over_MN;
                y[INDEX(M - hid, 0, N)] = tmp_down.x * two_over_MN;

                tmp1 = complexAdd(V[INDEX(hid, halfN, halfN + 1)], V[INDEX(M - hid, halfN, halfN + 1)]);
                tmp2 = complexSubtract(V[INDEX(hid, halfN, halfN + 1)], V[INDEX(M - hid, halfN, halfN + 1)]);
                tmp_up.x = expkM[hid].x * tmp1.x - expkM[hid].y * tmp2.y;
                tmp_up.y = expkM[hid].x * tmp1.y + expkM[hid].y * tmp2.x;
                tmp_down.x = -expkM[hid].y * tmp1.x - expkM[hid].x * tmp2.y;
                tmp_down.y = -expkM[hid].y * tmp1.y + expkM[hid].x * tmp2.x;
                y[INDEX(hid, halfN, N)] = RealPartOfMul(expkN[halfN], tmp_up) * two_over_MN;
                y[INDEX(M - hid, halfN, N)] = RealPartOfMul(expkN[halfN], tmp_down) * two_over_MN;
                break;
            }

            case 3: {
                ComplexType<T> tmp1, tmp2, tmp_up, tmp_down;
                tmp1 = complexAdd(V[INDEX(hid, wid, halfN + 1)], V[INDEX(M - hid, wid, halfN + 1)]);
                tmp2 = complexSubtract(V[INDEX(hid, wid, halfN + 1)], V[INDEX(M - hid, wid, halfN + 1)]);
                tmp_up.x = expkM[hid].x * tmp1.x - expkM[hid].y * tmp2.y;
                tmp_up.y = expkM[hid].x * tmp1.y + expkM[hid].y * tmp2.x;
                tmp_down.x = -expkM[hid].y * tmp1.x - expkM[hid].x * tmp2.y;
                tmp_down.y = -expkM[hid].y * tmp1.y + expkM[hid].x * tmp2.x;
                y[INDEX(hid, wid, N)] = RealPartOfMul(expkN[wid], tmp_up) * two_over_MN;
                y[INDEX(M - hid, wid, N)] = RealPartOfMul(expkN[wid], tmp_down) * two_over_MN;
                y[INDEX(hid, N - wid, N)] = -ImaginaryPartOfMul(expkN[wid], tmp_up) * two_over_MN;
                y[INDEX(M - hid, N - wid, N)] = -ImaginaryPartOfMul(expkN[wid], tmp_down) * two_over_MN;
                break;
            }

            default:
                break;
        }
    }
}

template <typename T>
void dct2dPostprocessCudaLauncher(
    const T *x, T *y, const int M, const int N, const T *__restrict__ expkM, const T *__restrict__ expkN) {
    dim3 gridSize((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    dct2dPostprocess<T, ComplexType<T>><<<gridSize, blockSize, 0, stream>>>((ComplexType<T> *)x,
                                                                            y,
                                                                            M,
                                                                            N,
                                                                            M / 2,
                                                                            N / 2,
                                                                            (T)(2. / (M * N)),
                                                                            (T)(4. / (M * N)),
                                                                            (ComplexType<T> *)expkM,
                                                                            (ComplexType<T> *)expkN);
}

// idct2_fft2
template <typename T, typename TComplex>
__global__ void __launch_bounds__(TPB *TPB, 4) idct2_fft2Preprocess(const T *input,
                                                                    TComplex *output,
                                                                    const int M,
                                                                    const int N,
                                                                    const int halfM,
                                                                    const int halfN,
                                                                    const TComplex *__restrict__ expkM,
                                                                    const TComplex *__restrict__ expkN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < halfM && wid < halfN) {
        int cond = ((hid != 0) << 1) | (wid != 0);
        switch (cond) {
            case 0: {
                T tmp1;
                TComplex tmp_up;

                output[0].x = input[0];
                output[0].y = 0;

                tmp1 = input[halfN];
                tmp_up.x = tmp1;
                tmp_up.y = tmp1;
                output[halfN] = complexConj(complexMul(expkN[halfN], tmp_up));

                tmp1 = input[INDEX(halfM, 0, N)];
                tmp_up.x = tmp1;
                tmp_up.y = tmp1;
                output[INDEX(halfM, 0, halfN + 1)] = complexConj(complexMul(expkM[halfM], tmp_up));

                tmp1 = input[INDEX(halfM, halfN, N)];
                tmp_up.x = 0;
                tmp_up.y = 2 * tmp1;
                output[INDEX(halfM, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[halfM], expkN[halfN]), tmp_up));
                break;
            }

            case 1: {
                TComplex tmp_up;
                tmp_up.x = input[wid];
                tmp_up.y = input[N - wid];
                output[wid] = complexConj(complexMul(expkN[wid], tmp_up));

                T tmp1 = input[INDEX(halfM, wid, N)];
                T tmp2 = input[INDEX(halfM, N - wid, N)];
                tmp_up.x = tmp1 - tmp2;
                tmp_up.y = tmp1 + tmp2;
                output[INDEX(halfM, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[halfM], expkN[wid]), tmp_up));
                break;
            }

            case 2: {
                T tmp1, tmp3;
                TComplex tmp_up, tmp_down;

                tmp1 = input[INDEX(hid, 0, N)];
                tmp3 = input[INDEX(M - hid, 0, N)];
                tmp_up.x = tmp1;
                tmp_up.y = tmp3;
                tmp_down.x = tmp3;
                tmp_down.y = tmp1;

                output[INDEX(hid, 0, halfN + 1)] = complexConj(complexMul(expkM[hid], tmp_up));
                output[INDEX(M - hid, 0, halfN + 1)] = complexConj(complexMul(expkM[M - hid], tmp_down));

                tmp1 = input[INDEX(hid, halfN, N)];
                tmp3 = input[INDEX(M - hid, halfN, N)];
                tmp_up.x = tmp1 - tmp3;
                tmp_up.y = tmp3 + tmp1;
                tmp_down.x = tmp3 - tmp1;
                tmp_down.y = tmp1 + tmp3;

                output[INDEX(hid, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[hid], expkN[halfN]), tmp_up));
                output[INDEX(M - hid, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[M - hid], expkN[halfN]), tmp_down));
                break;
            }

            case 3: {
                T tmp1 = input[INDEX(hid, wid, N)];
                T tmp2 = input[INDEX(hid, N - wid, N)];
                T tmp3 = input[INDEX(M - hid, wid, N)];
                T tmp4 = input[INDEX(M - hid, N - wid, N)];
                TComplex tmp_up, tmp_down;
                tmp_up.x = tmp1 - tmp4;
                tmp_up.y = tmp3 + tmp2;
                tmp_down.x = tmp3 - tmp2;
                tmp_down.y = tmp1 + tmp4;

                output[INDEX(hid, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[hid], expkN[wid]), tmp_up));
                output[INDEX(M - hid, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[M - hid], expkN[wid]), tmp_down));
                break;
            }

            default:
                break;
        }
    }
}

template <typename T>
void idct2_fft2PreprocessCudaLauncher(
    const T *x, T *y, const int M, const int N, const T *__restrict__ expkM, const T *__restrict__ expkN) {
    dim3 gridSize((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    idct2_fft2Preprocess<T, ComplexType<T>><<<gridSize, blockSize, 0, stream>>>(
        x, (ComplexType<T> *)y, M, N, M / 2, N / 2, (ComplexType<T> *)expkM, (ComplexType<T> *)expkN);
}

template <typename T>
__global__ void idct2_fft2Postprocess(const T *x, T *y, const int M, const int N, const int halfN, const int MN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N) {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond) {
            case 0:
                index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
                break;
            case 1:
                index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
                break;
            case 2:
                index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
                break;
            case 3:
                index = INDEX(hid << 1, wid << 1, N);
                break;
            default:
                break;
        }
        y[index] = x[INDEX(hid, wid, N)] * MN;
    }
}

template <typename T>
void idct2_fft2PostprocessCudaLauncher(const T *x, T *y, const int M, const int N) {
    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    idct2_fft2Postprocess<T><<<gridSize, blockSize, 0, stream>>>(x, y, M, N, N / 2, M * N);
}

// idct_idxst
// Adpated from idct2d_preprocess(). The only change is the reordered input
// if (wid != 0)
//     new_input[hid][wid] = input[hid][N - wid];
// else
//     new_input[hid][0] = 0
template <typename T, typename TComplex>
__global__ void __launch_bounds__(TPB *TPB, 4) idct_idxstPreprocess(const T *input,
                                                                    TComplex *output,
                                                                    const int M,
                                                                    const int N,
                                                                    const int halfM,
                                                                    const int halfN,
                                                                    const TComplex *__restrict__ expkM,
                                                                    const TComplex *__restrict__ expkN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < halfM && wid < halfN) {
        int cond = ((hid != 0) << 1) | (wid != 0);
        switch (cond) {
            case 0: {
                T tmp1;
                TComplex tmp_up;

                output[0].x = 0;
                output[0].y = 0;

                tmp1 = input[halfN];
                tmp_up.x = tmp1;
                tmp_up.y = tmp1;
                output[halfN] = complexConj(complexMul(expkN[halfN], tmp_up));

                output[INDEX(halfM, 0, halfN + 1)].x = 0;
                output[INDEX(halfM, 0, halfN + 1)].y = 0;

                tmp1 = input[INDEX(halfM, halfN, N)];
                tmp_up.x = 0;
                tmp_up.y = 2 * tmp1;
                output[INDEX(halfM, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[halfM], expkN[halfN]), tmp_up));
                break;
            }

            case 1: {
                TComplex tmp_up;
                tmp_up.x = input[N - wid];
                tmp_up.y = input[wid];
                output[wid] = complexConj(complexMul(expkN[wid], tmp_up));

                T tmp1 = input[INDEX(halfM, N - wid, N)];
                T tmp2 = input[INDEX(halfM, wid, N)];
                tmp_up.x = tmp1 - tmp2;
                tmp_up.y = tmp1 + tmp2;
                output[INDEX(halfM, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[halfM], expkN[wid]), tmp_up));
                break;
            }

            case 2: {
                T tmp1, tmp3;
                TComplex tmp_up, tmp_down;

                output[INDEX(hid, 0, halfN + 1)].x = 0;
                output[INDEX(hid, 0, halfN + 1)].y = 0;
                output[INDEX(M - hid, 0, halfN + 1)].x = 0;
                output[INDEX(M - hid, 0, halfN + 1)].y = 0;

                tmp1 = input[INDEX(hid, halfN, N)];
                tmp3 = input[INDEX(M - hid, halfN, N)];
                tmp_up.x = tmp1 - tmp3;
                tmp_up.y = tmp3 + tmp1;
                tmp_down.x = tmp3 - tmp1;
                tmp_down.y = tmp1 + tmp3;

                output[INDEX(hid, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[hid], expkN[halfN]), tmp_up));
                output[INDEX(M - hid, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[M - hid], expkN[halfN]), tmp_down));
                break;
            }

            case 3: {
                T tmp1 = input[INDEX(hid, N - wid, N)];
                T tmp2 = input[INDEX(hid, wid, N)];
                T tmp3 = input[INDEX(M - hid, N - wid, N)];
                T tmp4 = input[INDEX(M - hid, wid, N)];
                TComplex tmp_up, tmp_down;
                tmp_up.x = tmp1 - tmp4;
                tmp_up.y = tmp3 + tmp2;
                tmp_down.x = tmp3 - tmp2;
                tmp_down.y = tmp1 + tmp4;

                output[INDEX(hid, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[hid], expkN[wid]), tmp_up));
                output[INDEX(M - hid, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[M - hid], expkN[wid]), tmp_down));
                break;
            }

            default:
                break;
        }
    }
}

template <typename T>
void idct_idxstPreprocessCudaLauncher(
    const T *x, T *y, const int M, const int N, const T *__restrict__ expkM, const T *__restrict__ expkN) {
    dim3 gridSize((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    idct_idxstPreprocess<T, ComplexType<T>><<<gridSize, blockSize, 0, stream>>>(
        x, (ComplexType<T> *)y, M, N, M / 2, N / 2, (ComplexType<T> *)expkM, (ComplexType<T> *)expkN);
}

// Adpated from idct2d_postprocess() with changes on sign and scale
// if (wid % 2 == 1)
//     new_output[hid][wid] = -output[hid][wid];
// else
//     new_output[hid][wid] = output[hid][wid];
template <typename T>
__global__ void idct_idxstPostprocess(const T *x, T *y, const int M, const int N, const int halfN, const int MN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N) {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond) {
            case 0:
                index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
                y[index] = -x[INDEX(hid, wid, N)] * MN;
                break;
            case 1:
                index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
                y[index] = x[INDEX(hid, wid, N)] * MN;
                break;
            case 2:
                index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
                y[index] = -x[INDEX(hid, wid, N)] * MN;
                break;
            case 3:
                index = INDEX(hid << 1, wid << 1, N);
                y[index] = x[INDEX(hid, wid, N)] * MN;
                break;
            default:
                break;
        }
    }
}

template <typename T>
void idct_idxstPostprocessCudaLauncher(const T *x, T *y, const int M, const int N) {
    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    idct_idxstPostprocess<T><<<gridSize, blockSize, 0, stream>>>(x, y, M, N, N / 2, M * N);
}

// idxst_idct
// Adpated from idct2d_preprocess(). The only change is the reordered input
// if (hid != 0)
//     new_input[hid][wid] = input[M - hid][wid];
// else
//     new_input[0][wid] = 0
template <typename T, typename TComplex>
__global__ void __launch_bounds__(TPB *TPB, 4) idxst_idctPreprocess(const T *input,
                                                                    TComplex *output,
                                                                    const int M,
                                                                    const int N,
                                                                    const int halfM,
                                                                    const int halfN,
                                                                    const TComplex *__restrict__ expkM,
                                                                    const TComplex *__restrict__ expkN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < halfM && wid < halfN) {
        int cond = ((hid != 0) << 1) | (wid != 0);
        switch (cond) {
            case 0: {
                T tmp1;
                TComplex tmp_up;

                output[0].x = 0;
                output[0].y = 0;

                output[halfN].x = 0;
                output[halfN].y = 0;

                tmp1 = input[INDEX(halfM, 0, N)];
                tmp_up.x = tmp1;
                tmp_up.y = tmp1;
                output[INDEX(halfM, 0, halfN + 1)] = complexConj(complexMul(expkM[halfM], tmp_up));

                tmp1 = input[INDEX(halfM, halfN, N)];
                tmp_up.x = 0;
                tmp_up.y = 2 * tmp1;
                output[INDEX(halfM, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[halfM], expkN[halfN]), tmp_up));
                break;
            }

            case 1: {
                output[wid].x = 0;
                output[wid].y = 0;

                TComplex tmp_up;
                T tmp1 = input[INDEX(halfM, wid, N)];
                T tmp2 = input[INDEX(halfM, N - wid, N)];
                tmp_up.x = tmp1 - tmp2;
                tmp_up.y = tmp1 + tmp2;
                output[INDEX(halfM, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[halfM], expkN[wid]), tmp_up));
                break;
            }

            case 2: {
                T tmp1, tmp3;
                TComplex tmp_up, tmp_down;

                tmp1 = input[INDEX(M - hid, 0, N)];
                tmp3 = input[INDEX(hid, 0, N)];
                tmp_up.x = tmp1;
                tmp_up.y = tmp3;
                tmp_down.x = tmp3;
                tmp_down.y = tmp1;

                output[INDEX(hid, 0, halfN + 1)] = complexConj(complexMul(expkM[hid], tmp_up));
                output[INDEX(M - hid, 0, halfN + 1)] = complexConj(complexMul(expkM[M - hid], tmp_down));

                tmp1 = input[INDEX(M - hid, halfN, N)];
                tmp3 = input[INDEX(hid, halfN, N)];
                tmp_up.x = tmp1 - tmp3;
                tmp_up.y = tmp3 + tmp1;
                tmp_down.x = tmp3 - tmp1;
                tmp_down.y = tmp1 + tmp3;

                output[INDEX(hid, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[hid], expkN[halfN]), tmp_up));
                output[INDEX(M - hid, halfN, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[M - hid], expkN[halfN]), tmp_down));
                break;
            }

            case 3: {
                T tmp1 = input[INDEX(M - hid, wid, N)];
                T tmp2 = input[INDEX(M - hid, N - wid, N)];
                T tmp3 = input[INDEX(hid, wid, N)];
                T tmp4 = input[INDEX(hid, N - wid, N)];
                TComplex tmp_up, tmp_down;
                tmp_up.x = tmp1 - tmp4;
                tmp_up.y = tmp3 + tmp2;
                tmp_down.x = tmp3 - tmp2;
                tmp_down.y = tmp1 + tmp4;

                output[INDEX(hid, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[hid], expkN[wid]), tmp_up));
                output[INDEX(M - hid, wid, halfN + 1)] =
                    complexConj(complexMul(complexMul(expkM[M - hid], expkN[wid]), tmp_down));
                break;
            }

            default:
                break;
        }
    }
}

template <typename T>
void idxst_idctPreprocessCudaLauncher(
    const T *x, T *y, const int M, const int N, const T *__restrict__ expkM, const T *__restrict__ expkN) {
    dim3 gridSize((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    idxst_idctPreprocess<T, ComplexType<T>><<<gridSize, blockSize, 0, stream>>>(
        x, (ComplexType<T> *)y, M, N, M / 2, N / 2, (ComplexType<T> *)expkM, (ComplexType<T> *)expkN);
}

// Adpated from idct2d_postprocess() with changes on sign and scale
// if (hid % 2 == 1)
//     new_output[hid][wid] = -output[hid][wid];
// else
//     new_output[hid][wid] = output[hid][wid];
template <typename T>
__global__ void idxst_idctPostprocess(const T *x, T *y, const int M, const int N, const int halfN, const int MN) {
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N) {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond) {
            case 0:
                index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
                y[index] = -x[INDEX(hid, wid, N)] * MN;
                break;
            case 1:
                index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
                y[index] = -x[INDEX(hid, wid, N)] * MN;
                break;
            case 2:
                index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
                y[index] = x[INDEX(hid, wid, N)] * MN;
                break;
            case 3:
                index = INDEX(hid << 1, wid << 1, N);
                y[index] = x[INDEX(hid, wid, N)] * MN;
                break;
            default:
                break;
        }
    }
}

template <typename T>
void idxst_idctPostprocessCudaLauncher(const T *x, T *y, const int M, const int N) {
    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    auto stream = at::cuda::getCurrentCUDAStream();
    idxst_idctPostprocess<T><<<gridSize, blockSize, 0, stream>>>(x, y, M, N, N / 2, M * N);
}

// launch

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

void dct2_fft2_forward_cuda(at::Tensor x, at::Tensor expkM, at::Tensor expkN, at::Tensor out, at::Tensor buf) {
    hipSetDevice(x.get_device());
    CHECK_INPUT(x);
    CHECK_INPUT(expkM);
    CHECK_INPUT(expkN);
    CHECK_INPUT(out);
    CHECK_INPUT(buf);

    auto N = x.size(-1);
    auto M = x.numel() / N;

    dct2dPreprocessCudaLauncher<float>(x.data_ptr<float>(), out.data_ptr<float>(), M, N);

    buf = at::view_as_real(at::fft_rfft2(out, c10::nullopt, {-2, -1}, "backward")).contiguous();

    dct2dPostprocessCudaLauncher<float>(
        buf.data_ptr<float>(), out.data_ptr<float>(), M, N, expkM.data_ptr<float>(), expkN.data_ptr<float>());
}

void idct2_fft2_forward_cuda(at::Tensor x, at::Tensor expkM, at::Tensor expkN, at::Tensor out, at::Tensor buf) {
    hipSetDevice(x.get_device());
    CHECK_INPUT(x);
    CHECK_INPUT(expkM);
    CHECK_INPUT(expkN);
    CHECK_INPUT(out);
    CHECK_INPUT(buf);

    auto N = x.size(-1);
    auto M = x.numel() / N;

    idct2_fft2PreprocessCudaLauncher<float>(
        x.data_ptr<float>(), buf.data_ptr<float>(), M, N, expkM.data_ptr<float>(), expkN.data_ptr<float>());

    auto y = at::fft_irfft2(at::view_as_complex(buf), {{M, N}}, {-2, -1}, "backward").contiguous();

    idct2_fft2PostprocessCudaLauncher<float>(y.data_ptr<float>(), out.data_ptr<float>(), M, N);
}

void idct_idxst_forward_cuda(at::Tensor x, at::Tensor expkM, at::Tensor expkN, at::Tensor out, at::Tensor buf) {
    hipSetDevice(x.get_device());
    CHECK_INPUT(x);
    CHECK_INPUT(expkM);
    CHECK_INPUT(expkN);
    CHECK_INPUT(out);
    CHECK_INPUT(buf);

    auto N = x.size(-1);
    auto M = x.numel() / N;

    idct_idxstPreprocessCudaLauncher<float>(
        x.data_ptr<float>(), buf.data_ptr<float>(), M, N, expkM.data_ptr<float>(), expkN.data_ptr<float>());

    auto y = at::fft_irfft2(at::view_as_complex(buf), {{M, N}}, {-2, -1}, "backward").contiguous();

    idct_idxstPostprocessCudaLauncher<float>(y.data_ptr<float>(), out.data_ptr<float>(), M, N);
}

void idxst_idct_forward_cuda(at::Tensor x, at::Tensor expkM, at::Tensor expkN, at::Tensor out, at::Tensor buf) {
    hipSetDevice(x.get_device());
    CHECK_INPUT(x);
    CHECK_INPUT(expkM);
    CHECK_INPUT(expkN);
    CHECK_INPUT(out);
    CHECK_INPUT(buf);

    auto N = x.size(-1);
    auto M = x.numel() / N;

    idxst_idctPreprocessCudaLauncher<float>(
        x.data_ptr<float>(), buf.data_ptr<float>(), M, N, expkM.data_ptr<float>(), expkN.data_ptr<float>());

    auto y = at::fft_irfft2(at::view_as_complex(buf), {{M, N}}, {-2, -1}, "backward").contiguous();

    idxst_idctPostprocessCudaLauncher<float>(y.data_ptr<float>(), out.data_ptr<float>(), M, N);
}